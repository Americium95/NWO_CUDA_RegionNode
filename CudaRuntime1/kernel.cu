#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h> 

//메모리 쓰기
hipError_t menCopyWithCuda(const float4* a, unsigned int arraySize);
//거리연산
hipError_t disFilterWithCuda(float *c, const float4 start, unsigned int size);
int memFreeWithCuda();

//커널함수 정의
__global__ void disFilterKernel(float *c,float4 start, const float4 *a, unsigned int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;  // 남는 스레드 무시
    if (abs(a[i].x - start.x) + abs(a[i].y - start.y) < 5)
    {
        c[i] = abs(a[i].z - start.z + (a[i].x - start.x) * 2560) + abs(a[i].w - start.w - (a[i].y - start.y) * 2560);
    }
    else
        c[i] = 100000;
}


//CUDA디버깅용
int main()
{
    const int arraySize = 5;
    const float4 a[arraySize] = { {1, 2, 3}, {4, 5, 6}, {7, 8, 9}, {10, 11, 12}, {13, 14, 15} };
    const float4 b[arraySize] = { {15,14,13}, {12,11,10}, {9,8,7}, {6,5,4}, {3,2,1} };
    float c[arraySize] = { -1 };

    // Add vectors in parallel.
    hipError_t cudaStatus = menCopyWithCuda(a,arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    cudaStatus = disFilterWithCuda(c,b[0], arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // 결과 출력
    for (int i = 0; i < arraySize; ++i) {
        printf("%d:{%.1f, %.1f, %.1f} + {%.1f, %.1f, %.1f} = {%.1f}\n", i,
            a[i].x, a[i].y, a[i].z,
            b[0].x, b[0].y, b[0].z,
            c[i]);
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}


// Perform GPU computation
float4* dev_a;
float* dev_c;
hipError_t cudaStatus;

extern "C" __declspec(dllexport) int cudaMemCopy(float4* a, int arraySize)
{
    menCopyWithCuda(a,arraySize);
    return 0;
}


extern "C" __declspec(dllexport) float* exportCppFunctionAdd(float* dst, float4 start, int arraySize)
{
    // Perform GPU computation
    hipError_t cudaStatus = disFilterWithCuda(dst, start, arraySize);
    return dst;
    /*
    // Error checking (optional)
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "disFilterWithCuda failed: %s\n", hipGetErrorString(cudaStatus));
        // Returning an error code to indicate failure
        return -1;
    }

    // Return the required value
    return 88888888;
    */
}

extern "C" __declspec(dllexport) int cudaMemFree(float4* a, int arraySize)
{
    memFreeWithCuda();
    return 0;
}


hipError_t menCopyWithCuda(const float4* a, unsigned int arraySize)
{
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&dev_a, arraySize * sizeof(float4));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!0");
        hipFree(dev_a);
        return cudaStatus;
    }


    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, arraySize * sizeof(float4));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!1");
        memFreeWithCuda();
    }

    // gpu버퍼로 입력
    cudaStatus = hipMemcpy(dev_a, a, arraySize * sizeof(float4), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!2");
        memFreeWithCuda();
        return cudaStatus;
    }

    return cudaStatus;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t disFilterWithCuda(float *c, const float4 start, unsigned int size)
{
    unsigned int threadsPerBlock = 256;
    unsigned int blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    // 커널실행
    disFilterKernel << <blocks, threadsPerBlock >> > (dev_c, start, dev_a, size);
    //disFilterKernel << <1, size >> > (dev_c, start, dev_a);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        memFreeWithCuda();
        return cudaStatus;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        memFreeWithCuda();
    }

    // 출력결과 메모리로 복사
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!3");
        memFreeWithCuda();
        return cudaStatus;
    }

    //메모리 해제
//Error:
    //hipFree(dev_c);
    //hipFree(dev_a);

    return cudaStatus;
}

int memFreeWithCuda()
{
    hipFree(dev_c);
    hipFree(dev_a);
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
/*hipError_t disFilterWithCuda(float4* c, const float4 start, const float4* a, unsigned int size)
{
    float4* dev_c;
    float4* dev_a;

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(float4));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float4));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // gpu버퍼로 입력
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float4), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // 커널실행
    disFilterKernel << <1, size >> > (dev_c, start, dev_a);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // 출력결과 메모리로 복사
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(float4), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //메모리 해제
Error:
    hipFree(dev_c);
    hipFree(dev_a);

    return cudaStatus;
}*/